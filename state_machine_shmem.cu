#include "hip/hip_runtime.h"
#include "state_machine_shmem.cuh"

#include <vector>
#include "common/utils.hpp"
#include "kmp_cpu.hpp"
#include "state_machine_cpu.hpp"

// This value should be several times larger than pattern_length.
static constexpr int match_length_per_thread = 128;

static constexpr int block_size = 128;

static __host__ __device__ int ceil_div(int x, int y) {
    return (x - 1) / y + 1;
}

// Get arr[idx], where arr is the compact form of the gene sequence.
template<class IdxType>
static __host__ __device__ inline char get(const char *arr, IdxType idx) {
    return (arr[idx>>2] >> ((idx & 0x3) << 1)) & 0x3;
}

__device__ static void state_machine_search(
    const char *text, const int text_length, int match_start, int match_end,
    const int16_t pattern_length, int *output, int *output_cnt, const int max_output_cnt,
    const int16_t (*jump_table)[4]
) {
    if (match_start < match_end) {
        int i = match_start;
        int curr_state = 0;
        char packed_text = text[i >> 2] >> ((i & 0x3) << 1);
        for (; i < match_end; i++, packed_text >>=2) {
            if (!(i & 0x3)) {
                packed_text = text[i >> 2];
            }
            curr_state = jump_table[curr_state][packed_text & 0x3];
            if (curr_state == pattern_length) {
                int outputIdx = atomicAdd(output_cnt, 1);
                if (outputIdx >= max_output_cnt) {
                    return;
                }
                output[outputIdx] = i - pattern_length + 1;
            }
        }
    }
}

__global__ void state_machine_search_shmem_kernel(
    const char *text, const int text_length, const int16_t pattern_length,
    int *output, int *output_cnt, const int max_output_cnt, int16_t (*jump_table)[4]
) {
    extern __shared__ int16_t shared_jump_table[][4];

    int global_index = blockIdx.x * blockDim.x + threadIdx.x;
    int match_start = (match_length_per_thread - (pattern_length - 1)) * global_index;
    int match_end = match_start + match_length_per_thread;
    match_start = min(match_start, text_length);
    match_end = min(match_end, text_length);

    // Initialize shared memory.
    for (int i = threadIdx.x; i < 4 * (pattern_length + 1); i += blockDim.x) {
        shared_jump_table[i>>2][i&0x3] = jump_table[i>>2][i&0x3];
    }
    __syncthreads();

    state_machine_search(
        text, text_length, match_start, match_end,
        pattern_length, output, output_cnt, max_output_cnt, shared_jump_table
    );
}

int state_machine_search_shmem(
    const char *text, int text_length, const char *pattern, int16_t pattern_length,
    int *output, int max_output_cnt, int16_t *fail
) {
    if (match_length_per_thread <= pattern_length) {
        fprintf(stderr, "match_length_per_thread should be larger than pattern_length");
        exit(1);
    }

    timer_start("Computing state machine jump table on the CPU");
    std::vector<int16_t> jump_table(4 * (pattern_length+1));
    get_fail(pattern, pattern_length, fail);
    build_state_machine(
        reinterpret_cast<int16_t (*)[4]>(jump_table.data()),
        pattern, fail, pattern_length
    );
    timer_stop();

    // Array sizes, in bytes.
    int text_size = ceil_div(text_length * sizeof(char), 4);
    int jump_table_size = sizeof(int16_t) * (pattern_length + 1) * 4;
    int output_size = sizeof(int) * max_output_cnt;

    timer_start("Allocating GPU memory");
    char *text_device;
    int *output_device;
    int16_t *jump_table_device;
    int *output_cnt_device;
    THROW_IF_ERROR(hipMalloc((void **)&text_device, text_size));
    THROW_IF_ERROR(hipMalloc((void **)&output_device, output_size));
    THROW_IF_ERROR(hipMalloc((void **)&jump_table_device, jump_table_size));
    THROW_IF_ERROR(hipMalloc((void **)&output_cnt_device, sizeof(int)));
    timer_stop();

    timer_start("Copying inputs to the GPU");
    THROW_IF_ERROR(hipMemcpy(text_device, text, text_size, hipMemcpyHostToDevice));
    THROW_IF_ERROR(hipMemcpy(jump_table_device, jump_table.data(), jump_table_size, hipMemcpyHostToDevice));
    THROW_IF_ERROR(hipMemset(output_cnt_device, 0, sizeof(int)));
    timer_stop();

    // Prepare to launch the kernel.

    int num_blocks = ceil_div(
        text_length - (pattern_length - 1),
        block_size * (match_length_per_thread - (pattern_length - 1))
    );

    timer_start("Performing state machine search on the GPU");
    state_machine_search_shmem_kernel<<<num_blocks, block_size, jump_table_size>>>(
        text_device, text_length, pattern_length,
        output_device, output_cnt_device, max_output_cnt, reinterpret_cast<int16_t (*)[4]>(jump_table_device)
    );
    THROW_IF_ERROR(hipDeviceSynchronize());
    timer_stop();

    timer_start("Copying output memory to the CPU");
    int output_cnt;
    THROW_IF_ERROR(
        hipMemcpy(&output_cnt, output_cnt_device, sizeof(int), hipMemcpyDeviceToHost)
    );
    output_cnt = min(output_cnt, max_output_cnt);  // This counter may overflow.
    THROW_IF_ERROR(hipMemcpy(output, output_device, output_size, hipMemcpyDeviceToHost));
    timer_stop();

    timer_start("Freeing GPU memory");
    hipFree(text_device);
    hipFree(output_device);
    hipFree(jump_table_device);
    hipFree(output_cnt_device);
    timer_stop();
    return output_cnt;
}
