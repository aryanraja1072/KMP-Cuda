#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int device = 0; device < deviceCount; ++device)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        printf("Device %d supports mapping host memory:  ", device);
        if (deviceProp.canMapHostMemory)
            printf("Yes\n");
        else
            printf("No\n");
    }

    return 0;
}

